#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#define _USE_MATH_DEFINES
#include <math.h>
#include <vector>
#include <utility>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


//#define dd double

__device__
const double G = 6.674e-11;
__device__
const double dt = 0.001;

struct MatPoint {
    double x;
    double y;
    double vx;
    double vy;
    double m;
};

__global__
void calcForce(double* X, MatPoint* device_points, int num_of_points, int partition_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int start = i * partition_size;
    int end = (i + 1) * partition_size;
    //printf("%d-%d\n", start, end);
    for (int i = start; i < end; i++) {
        //for (unsigned i = 0; i < num_of_points; i += 1) {
            double sum_x = 0;
            double sum_y = 0;
            double x_i = static_cast<MatPoint>(device_points[i]).x;
            double y_i = static_cast<MatPoint>(device_points[i]).y;
            double m_i = static_cast<MatPoint>(device_points[i]).m;
            //printf("%f-%f-%f\n", x_i, y_i, m_i);
            for (unsigned j = 0; j < num_of_points; ++j) {
                double x_j = static_cast<MatPoint>(device_points[j]).x;
                double y_j = static_cast<MatPoint>(device_points[j]).y;
                double m_j = static_cast<MatPoint>(device_points[j]).m;
                if (i == j) {
                    continue;
                }

                double dist = sqrt(pow((x_j - x_i), 2) + pow((y_j - y_i), 2));
                sum_x += m_j * (x_j - x_i) / pow(dist, 3);
                sum_y += m_j * (y_j - y_i) / pow(dist, 3);
            }

            X[i * 2] = (G * m_i * sum_x);
            X[i * 2 + 1] = (G * m_i * sum_y);
            //printf("%f-%f\n", X[i*2], X[i*2+1]);
        //}
    }
}

__global__
void simulationStep_device(MatPoint* device_points, double* X, int num_of_points) {
    //printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
    for (unsigned i = 0; i < num_of_points; i += 1) {
        double x_i = static_cast<MatPoint>(device_points[i]).x;
        double y_i = static_cast<MatPoint>(device_points[i]).y;
        double vx_i = static_cast<MatPoint>(device_points[i]).vx;
        double vy_i = static_cast<MatPoint>(device_points[i]).vy;
        double m_i = static_cast<MatPoint>(device_points[i]).m;
        //printf("->->%f<-<-\n", static_cast<MatPoint>(device_points[i]).x);
        device_points[i].vx += X[i * 2] / m_i * dt;
        device_points[i].vy += X[i * 2 + 1] / m_i * dt;
        device_points[i].x += vx_i * dt;
        device_points[i].y += vy_i * dt;
        //printf("->->->->%f<-<-<-<-\n", static_cast<MatPoint>(device_points[i]).x-x_i);
        //printf("->->->->%f<-<-<-<-\n", vx_i);
    }
}

void read_file_thrust(thrust::device_vector<MatPoint>& points) {
    std::ifstream file("input.txt");
    double x, y, vx, vy, m;
    while (!file.eof()) {
        file >> x >> y >> vx >> vy >> m;
        points.push_back({ x, y, vx, vy, m });
        //printf("%f<->%f<->%f<->%f<->%f\n", x, y, vx, vy, m);
    }
}

void print_results(std::ofstream& file, MatPoint* points, int num_of_points) {
    // std::ofstream file("output.txt");
    for (int i = 0; i < num_of_points; i++) {
        file << points[i].x << "," << points[i].y << ", ";
    }
    file << "\n";
}

int main() {
    std::vector<MatPoint> points;
    thrust::device_vector<MatPoint> device_points;
    thrust::device_vector<MatPoint> device_points0;
    thrust::device_vector<double> results(1000);

    read_file_thrust(device_points);
    MatPoint* device_points_pointers = thrust::raw_pointer_cast(device_points.data());
    MatPoint* device_points_pointers0 = (MatPoint*)malloc(device_points.size() * sizeof(MatPoint));

    double* device_results_pointers = thrust::raw_pointer_cast(results.data());
    double* results_pointers;
    results_pointers = (double*)malloc(1000 * sizeof(double));
    int num_of_points = device_points.size();

    std::ofstream file("output.txt");
    file << "t,";
    for (unsigned i = 0; i < device_points.size(); ++i) {
        file << "x" << i + 1 << ",y" << i + 1 << ",";
    }
    file << "\n";

    int num_of_threads = 32;
    int partition_size = num_of_points / num_of_threads;
    double time_sum = 0;
    double t = 0;
    int st = 0;
    while (t < 100) {
        st++;
        printf("t=%f\n", t);
        clock_t start = clock();
        calcForce<<<1, num_of_threads>>>(device_results_pointers, device_points_pointers, num_of_points, partition_size);
        simulationStep_device<<<1, 1>>>(device_points_pointers, device_results_pointers, num_of_points);
        //hipMemcpy(results_pointers, device_results_pointers, 1000 * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(device_points_pointers0, device_points_pointers, num_of_points * sizeof(MatPoint), hipMemcpyDeviceToHost);
        clock_t end = clock();
        time_sum += ((double)(end - start)) / CLOCKS_PER_SEC;
        print_results(file, device_points_pointers0, num_of_points);
        t += dt;
    }
    printf("Num of threads: %d, Time taken: %.10f\n", num_of_threads, time_sum/st);
    
    return 0;
}
